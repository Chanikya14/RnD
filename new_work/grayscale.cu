#include "hip/hip_runtime.h"
#include "image_processing.h"

__global__ void rgbToGray(const uchar3 *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        uchar3 rgb = input[idx];
        output[idx] = 0.299f * rgb.x + 0.587f * rgb.y + 0.114f * rgb.z;
    }
}
